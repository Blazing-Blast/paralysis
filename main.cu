
#include <hip/hip_runtime.h>
#include <iostream>
#include <cinttypes>
#include <random>
#include <chrono>

constexpr uint_fast8_t totalTurns = 231;
constexpr uint64_t maxRuns = 1000000000ULL;
constexpr int threadCount = 1 << 10;
constexpr int blockCount = 1 << 10;

__always_inline int main_unthreaded();
__always_inline int main_threaded();
__global__ void roll_cuda(uint_fast8_t *ret, uint64_t seed);
__device__ static __always_inline uint64_t wyrand(uint64_t &seed);

int main()
{
    uint_fast8_t highest = 0;
    uint_fast8_t *results;

    std::random_device rd;

    std::chrono::time_point<std::chrono::high_resolution_clock> before = std::chrono::high_resolution_clock::now();

    const uint64_t seed = ((uint64_t)rd() << 32) | rd();
    
    hipMallocManaged(&results, blockCount * threadCount * sizeof(*results));

    roll_cuda<<<blockCount, threadCount>>>(results, seed);

    hipDeviceSynchronize();

    for (size_t i = 0; i < blockCount * threadCount; i++)
    {
        highest = std::max(highest, results[i]);
    }

    std::chrono::time_point<std::chrono::high_resolution_clock> after = std::chrono::high_resolution_clock::now();
    std::chrono::duration time = after - before;

    std::cout << "My record is: " << (int)highest << ".\nIt took me " << time.count()/1000000 << "ms.\n";
    return 0;
}

__global__ void roll_cuda(uint_fast8_t *results, uint64_t seed)
{
    uint_fast8_t highest = 0;

    uint_fast8_t current;
    seed += threadIdx.x + blockIdx.x * threadCount;

    const size_t runs = (maxRuns / (threadCount * blockCount))
                      + ((threadIdx.x + blockIdx.x * blockCount) < (maxRuns % (threadCount * blockCount)));
    for (size_t i = 0; i < runs; i++)
    {
        current = __popcll(wyrand(seed) & wyrand(seed));
        current += __popcll(wyrand(seed) & wyrand(seed));
        current += __popcll(wyrand(seed) & wyrand(seed));
        current += __popcll(wyrand(seed) & wyrand(seed) << (256 - (totalTurns % 256)));
        highest = max(highest, current);
    }
    results[threadIdx.x] = highest;
}

__device__ static __always_inline uint64_t wyrand(uint64_t &seed)
{
    seed += 0xa0761d6478bd642full;
    uint64_t A = seed,
             B = seed ^ 0xe7037ed1a0b428dbull;
    __uint128_t r = A;
    r *= B;
    A = (uint64_t)r;
    B = (uint64_t)(r >> 64);
    return A ^ B;
}